#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Color/ColorSmooth/ColorSmooth.hpp"
#include <stdio.h>

/******************************/
/*         CUDA DOUBLE        */
/******************************/

__device__ unsigned short process_cuda_d(const double startReal, const double startImag, unsigned short max_iters)  {
  double zReal = startReal;
  double zImag = startImag;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    double r2 = zReal * zReal;
    double i2 = zImag * zImag;
    zImag = 2.0f * zReal * zImag + startImag;
    zReal = r2 - i2 + startReal;
    if ( (r2 + i2) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}


__global__ void kernel_compute_cuda_d(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  double imag;
  double real;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    imag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    real = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH +x] = process_cuda_d(real, imag, max_iters);
	}
}


void compute_cuda_d(int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_d<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);
}



/******************************/
/*         CUDA DOUBLE2       */
/******************************/

__device__ unsigned short process_cuda_d2(double2 startValue, unsigned short max_iters)  {

  double2 startZ = startValue;

  double2 z1;
  z1.x = startZ.x;
  z1.y = startZ.y;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    double2 z2;
    z2.x = z1.x * z1.x;
    z2.y = z1.y * z1.y;

    z1.y = 2.0f * z1.x * z1.y + startZ.y;
    z1.x = z2.x - z2.y + startZ.x;
    if ( (z2.x + z2.y) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}

__global__ void kernel_compute_cuda_d2(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH + x] = process_cuda_d2(z, max_iters);
	}
}


void compute_cuda_d2(int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_d2<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}


/******************************/
/*CUDA DOUBLE WITHOUT PROCESS */
/******************************/

__global__ void kernel_compute_cuda_d2_wp(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;
  double2 z1;
  double2 z2;

  if ( (i<IMAGE_WIDTH) && (j<IMAGE_HEIGHT) ) {
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (j * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+ (i * zoom);

    z1.x = z.x;
    z1.y = z.y;

    device_value[j*IMAGE_WIDTH + i] = max_iters - 1;

    for (unsigned short counter = 0; counter < max_iters; counter++) {
      z2.x = z1.x * z1.x;
      z2.y = z1.y * z1.y;

      z1.y = 2.0f * z1.x * z1.y + z.y;
      z1.x = z2.x - z2.y + z.x;
      if ( (z2.x + z2.y) > 4.0f) {
        device_value[j*IMAGE_WIDTH + i] = counter;
        break;
      }
    }
	}
}

void compute_cuda_d2_wp (int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_d2_wp<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);
}



/******************************/
/*        CUDA SIMPLE         */
/******************************/
__device__ unsigned short process_cuda_s(const float startReal, const float startImag, unsigned short max_iters)  {
  float zReal = startReal;
  float zImag = startImag;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    float r2 = zReal * zReal;
    float i2 = zImag * zImag;
    zImag = 2.0f * zReal * zImag + startImag;
    zReal = r2 - i2 + startReal;
    if ( (r2 + i2) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}

__global__ void kernel_compute_cuda_s(float zoom, float offsetX, float offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  float imag;
  float real;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    imag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    real = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH +x] = process_cuda_s(real, imag, max_iters);
	}
}

void compute_cuda_s (int nthreads, float zoom, float offsetX, float offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_s<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}
