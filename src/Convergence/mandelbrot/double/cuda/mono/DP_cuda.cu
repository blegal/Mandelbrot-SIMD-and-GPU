#include "hip/hip_runtime.h"
/*
 *
 *
 *##############################################################################
 *
 *
 */
#if defined(__NVCC__) || defined(__HIPCC__)
/*
 *
 *
 *##############################################################################
 *
 *
 */
#include "DP_cuda.hpp"
#include <iostream>
/*
 *
 *
 *##############################################################################
 *
 *
 */
DP_cuda::DP_cuda() : Convergence("DP_cuda")
{
    fractal     = "mandelbrot";
    dataFormat  = "double";
    modeSIMD    = "none";
    modeOPENMP  = "disable";
    OTHER       = "CUDA";

    // Initialisation of HOST dataset

    host_mBuffer  = nullptr;
    host_mZoom    = 0.f;
    host_mOffsetX = 0.f;
    host_mOffsetY = 0.f;
    host_mWidth   = 0;
    host_mHeight  = 0;
    host_mIters   = 0;

    // Initialisation of GPU dataset

    host_mBuffer = nullptr;
    gpu_mBuffer  = nullptr;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
DP_cuda::DP_cuda(ColorMap* _colors, int _max_iters) : Convergence("DP_cuda") {
    colors      = _colors;
    max_iters   = _max_iters;

    fractal     = "mandelbrot";
    dataFormat  = "double";
    modeSIMD    = "none";
    modeOPENMP  = "disable";
    OTHER       = "CUDA";


    // Initialisation of HOST dataset

    host_mBuffer  = nullptr;
    host_mZoom    = 0.f;
    host_mOffsetX = 0.f;
    host_mOffsetY = 0.f;
    host_mWidth   = 0;
    host_mHeight  = 0;
    host_mIters   = 0;

    // Initialisation of GPU dataset

    host_mBuffer = nullptr;
    gpu_mBuffer  = nullptr;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
DP_cuda::~DP_cuda( )
{
    if( gpu_mBuffer != nullptr)
        hipFree( gpu_mBuffer );

    if( host_mBuffer != nullptr)
        free( host_mBuffer );
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__  void mandelbrot_sp
(
          int*   v_dat,    // le pointeur sur le tableau de sortie
    const double zoom,     // Le facteur de zoom
    const double offsetX,  // la coordonnée X
    const double offsetY,  // la coordonnée Y
    const int    width,    // la largeur de la fenetre
    const int    height,   // la hauteur de la fenetre
    const int    max_iters // le nombre maximum d'itération
);

/*
 *
 *
 *##############################################################################
 *
 *
 */
void DP_cuda::updateImage(
        const long double _zoom,
        const long double _offsetX,
        const long double _offsetY,
        const int IMAGE_WIDTH,
        const int IMAGE_HEIGHT,
        float* ptr)
{
    hipError_t err      = hipSuccess;
    const int IMAGE_SIZE = IMAGE_WIDTH * IMAGE_HEIGHT;

    const long double corner_x = _offsetX - (IMAGE_WIDTH  / 2.0f) * _zoom;
    const long double corner_y = _offsetY - (IMAGE_HEIGHT / 2.0f) * _zoom;

    if( host_mBuffer == nullptr )
    {

        host_mBuffer = (int32_t*)malloc( IMAGE_SIZE * sizeof(int32_t) );
        if (host_mBuffer == nullptr) {
            fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        err = hipMalloc((void **)&gpu_mBuffer, IMAGE_SIZE * sizeof(int32_t));
        if (err != hipSuccess)
        {
            fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }
    }

    // End or parameter updates
    dim3 DimGrid (IMAGE_WIDTH, IMAGE_HEIGHT); // 5000 thread blocks
    dim3 DimBlock(16, 16);

    mandelbrot_sp<<<DimGrid, DimBlock>>>(
        gpu_mBuffer,
        _zoom,
        corner_x,
        corner_y,
        IMAGE_WIDTH,
        IMAGE_HEIGHT,
        max_iters
    );

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch mandelbrot_sp kernel (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(host_mBuffer, gpu_mBuffer, IMAGE_SIZE * sizeof(int32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector gpu_c_pos from device to host (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }


    // On push back dans le buffer logiciel les resultats de calcul.

    for (int p = 0; p < IMAGE_SIZE; p += 1)
    {
        ptr[p] = host_mBuffer[p];
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
bool DP_cuda::is_valid()
{
    return true;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
#endif