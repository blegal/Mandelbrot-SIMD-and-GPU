#include "hip/hip_runtime.h"
/*
 *
 *
 *##############################################################################
 *
 *
 */
#include "SP_cuda.hpp"
#include <iostream>
/*
 *
 *
 *##############################################################################
 *
 *
 */
SP_cuda::SP_cuda() : Convergence("SP_CUDA")
{
    fractal     = "mandelbrot";
    dataFormat  = "float";
    modeSIMD    = "none";
    modeOPENMP  = "disable";
    OTHER       = "CUDA";

    // Initialisation of HOST dataset

    host_mBuffer  = nullptr;
    host_mZoom    = 0.f;
    host_mOffsetX = 0.f;
    host_mOffsetY = 0.f;
    host_mWidth   = 0;
    host_mHeight  = 0;
    host_mIters   = 0;

    // Initialisation of GPU dataset

    host_mBuffer = nullptr;
    gpu_mBuffer  = nullptr;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
SP_cuda::SP_cuda(ColorMap* _colors, int _max_iters) : Convergence("SP_CUDA") {
    colors      = _colors;
    max_iters   = _max_iters;

    fractal     = "mandelbrot";
    dataFormat  = "float";
    modeSIMD    = "none";
    modeOPENMP  = "disable";
    OTHER       = "CUDA";


    // Initialisation of HOST dataset

    host_mBuffer  = nullptr;
    host_mZoom    = 0.f;
    host_mOffsetX = 0.f;
    host_mOffsetY = 0.f;
    host_mWidth   = 0;
    host_mHeight  = 0;
    host_mIters   = 0;

    // Initialisation of GPU dataset

    host_mBuffer = nullptr;
    gpu_mBuffer  = nullptr;
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
SP_cuda::~SP_cuda( )
{
    if( gpu_mBuffer != nullptr)
        hipFree( gpu_mBuffer );

    if( host_mBuffer != nullptr)
        free( host_mBuffer );
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
extern __global__  void mandelbrot_sp
(
          int*  v_dat,    // le pointeur sur le tableau de sortie
    const float zoom,     // Le facteur de zoom
    const float offsetX,  // la coordonnée X
    const float offsetY,  // la coordonnée Y
    const int   width,    // la largeur de la fenetre
    const int   height,   // la hauteur de la fenetre
    const int   max_iters // le nombre maximum d'itération
);

/*
 *
 *
 *##############################################################################
 *
 *
 */
void SP_cuda::updateImage(
        const long double _zoom,
        const long double _offsetX,
        const long double _offsetY,
        const int IMAGE_WIDTH,
        const int IMAGE_HEIGHT,
        float* ptr)
{
    hipError_t err      = hipSuccess;
    const int IMAGE_SIZE = IMAGE_WIDTH * IMAGE_HEIGHT;

    const long double corner_x = _offsetX - (IMAGE_WIDTH  / 2.0f) * _zoom;
    const long double corner_y = _offsetY - (IMAGE_HEIGHT / 2.0f) * _zoom;

    if( host_mBuffer == nullptr )
    {

        host_mBuffer = (int32_t*)malloc( IMAGE_SIZE * sizeof(int32_t) );
        if (host_mBuffer == nullptr) {
            fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        err = hipMalloc((void **)&gpu_mBuffer, IMAGE_SIZE * sizeof(int32_t));
        if (err != hipSuccess)
        {
            fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }
    }

    // End or parameter updates
    dim3 DimGrid (IMAGE_WIDTH, IMAGE_HEIGHT); // 5000 thread blocks
    dim3 DimBlock(16, 16);

    mandelbrot_sp<<<DimGrid, DimBlock>>>(
        gpu_mBuffer,
        _zoom,
        corner_x,
        corner_y,
        IMAGE_WIDTH,
        IMAGE_HEIGHT,
        max_iters
    );

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch mandelbrot_sp kernel (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(host_mBuffer, gpu_mBuffer, IMAGE_SIZE * sizeof(int32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector gpu_c_pos from device to host (error code %s)!\n", hipGetErrorString(err));
        fprintf(stderr, "issue happned in file %s at line %d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }


    // On push back dans le buffer logiciel les resultats de calcul.

    for (int p = 0; p < IMAGE_SIZE; p += 1)
    {
        ptr[p] = host_mBuffer[p];
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
bool SP_cuda::is_valid()
{
    return true;
}
