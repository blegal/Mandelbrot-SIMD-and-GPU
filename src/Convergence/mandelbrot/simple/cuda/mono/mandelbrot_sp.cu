
#include <hip/hip_runtime.h>

__device__ unsigned short process_cuda_d(const double startReal, const double startImag, unsigned short max_iters)  {
  double zReal = startReal;
  double zImag = startImag;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    double r2 = zReal * zReal;
    double i2 = zImag * zImag;
    zImag = 2.0f * zReal * zImag + startImag;
    zReal = r2 - i2 + startReal;
    if ( (r2 + i2) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}


__global__ void kernel_compute_cuda_d(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  double imag;
  double real;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    imag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    real = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH +x] = process_cuda_d(real, imag, max_iters);
	}
}


void compute_cuda_d(int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_d<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);
}



/******************************/
/*         CUDA DOUBLE2       */
/******************************/

__device__ unsigned short process_cuda_d2(double2 startValue, unsigned short max_iters)  {

  double2 startZ = startValue;

  double2 z1;
  z1.x = startZ.x;
  z1.y = startZ.y;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    double2 z2;
    z2.x = z1.x * z1.x;
    z2.y = z1.y * z1.y;

    z1.y = 2.0f * z1.x * z1.y + startZ.y;
    z1.x = z2.x - z2.y + startZ.x;
    if ( (z2.x + z2.y) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}

__global__ void kernel_compute_cuda_d2(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH + x] = process_cuda_d2(z, max_iters);
	}
}


void compute_cuda_d2(int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_d2<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}


/******************************/
/*CUDA DOUBLE WITHOUT PROCESS */
/******************************/

__global__ void kernel_compute_cuda_d2_wp(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;
  double2 z1;
  double2 z2;

  if ( (i<IMAGE_WIDTH) && (j<IMAGE_HEIGHT) ) {
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (j * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+ (i * zoom);

    z1.x = z.x;
    z1.y = z.y;

    device_value[j*IMAGE_WIDTH + i] = max_iters - 1;

    for (unsigned short counter = 0; counter < max_iters; counter++) {
      z2.x = z1.x * z1.x;
      z2.y = z1.y * z1.y;

      z1.y = 2.0f * z1.x * z1.y + z.y;
      z1.x = z2.x - z2.y + z.x;
      if ( (z2.x + z2.y) > 4.0f) {
        device_value[j*IMAGE_WIDTH + i] = counter;
        break;
      }
    }
	}
}

void compute_cuda_d2_wp (int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute_cuda_d2_wp<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);
}



/******************************/
/*        CUDA SIMPLE         */
/******************************/
__device__ unsigned short process_cuda_s(const float startReal, const float startImag, unsigned short max_iters)  {
  float zReal = startReal;
  float zImag = startImag;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    float r2 = zReal * zReal;
    float i2 = zImag * zImag;
    zImag = 2.0f * zReal * zImag + startImag;
    zReal = r2 - i2 + startReal;
    if ( (r2 + i2) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}

__global__ void mandelbrot_sp
    (
              int*  v_dat,    // le pointeur sur le tableau de sortie
        const float zoom,     // Le facteur de zoom
        const float offsetX,  // la coordonnée X
        const float offsetY,  // la coordonnée Y
        const int   width,    // la largeur de la fenetre
        const int   height,   // la hauteur de la fenetre
        const int   max_iters // le nombre maximum d'itération
    )
{

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ( (x < width) && (y < height) )
    {
        const float imag  = offsetY  + (y * zoom);
        const float real  = offsetX + (x * zoom);
        v_dat[y*width +x] = process_cuda_s(real, imag, max_iters);
	}
}
